#include "hip/hip_runtime.h"
//---------------------------------*-C++-*-----------------------------------//
/*!
 * \file   MC/cuda_geometry/test/Mesh_Geometry_Tester.cu
 * \author Steven Hamilton
 * \date   Mon Dec 14 13:28:26 2015
 * \brief  Mesh_Geometry_Tester class definitions.
 * \note   Copyright (c) 2015 Oak Ridge National Laboratory, UT-Battelle, LLC.
 */
//---------------------------------------------------------------------------//

#include "gtest/Gtest_Functions.hh"
#include "utils/View_Field.hh"
#include "cuda_utils/Definitions.hh"
#include "cuda_utils/CudaDBC.hh"

#include "../Mesh_Geometry.hh"
#include "Mesh_Geometry_Tester.hh"

typedef profugus::geometry::cell_type       cell_type;
typedef profugus::geometry::matid_type      matid_type;
typedef cuda_utils::Space_Vector            Point;
typedef cuda_utils::Coordinates             Coords;
typedef cuda_profugus::Mesh_Geometry        Mesh_Geometry;
typedef cuda_profugus::Mesh_Geometry_DMM    Mesh_Geometry_DMM;

// Compute the matid for particle at each specified spatial location
__global__ void compute_matids_kernel(Mesh_Geometry   mesh,
                                      int             num_points,
                                      const Point    *points,
                                      int            *matids)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if( tid < num_points )
    {
        // Create and initialize state on each thread
        // We're only testing matids so direction doesn't matter
        cuda_profugus::Mesh_State state;
        Point dir = {1.0, 0.0, 0.0};
        mesh.initialize(points[tid],dir,state);

        // Get matid
        matids[tid] = mesh.matid(state);
    }
}

// Compute the distance to boundary
__global__ void distance_kernel(Mesh_Geometry   mesh,
                                int             num_points,
                                const Point    *points,
                                        const Point    *dirs,
                                        double         *distances,
                                        Coords         *next_ijk)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if( tid < num_points )
    {
        // Create and initialize state on each thread
        cuda_profugus::Mesh_State state;
        mesh.initialize(points[tid],dirs[tid],state);

        distances[tid] = mesh.distance_to_boundary(state);
        next_ijk[tid] = state.next_ijk;
    }
}

// Compute the distance to boundary
__global__ void move_to_surf_kernel(Mesh_Geometry   mesh,
                                    int             num_points,
                                    const Point    *points,
                                    const Point    *dirs,
                                    Coords         *ijk)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if( tid < num_points )
    {
        // Create and initialize state on each thread
        cuda_profugus::Mesh_State state;
        mesh.initialize(points[tid],dirs[tid],state);

        mesh.distance_to_boundary(state);
        mesh.move_to_surface(state);
        ijk[tid] = state.ijk;
    }
}

// Compute the distance to boundary
__global__ void reflect_kernel(Mesh_Geometry   mesh,
                               int             num_points,
                               const Point    *points,
                               const Point    *dirs_in,
                               int            *reflected,
                               Point          *dirs_out,
                               Coords         *ijk_out,
                               int            *exit_face,
                               int            *refl_face)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if( tid < num_points )
    {
        // Create and initialize state on each thread
        cuda_profugus::Mesh_State state;
        mesh.initialize(points[tid],dirs_in[tid],state);

        // Move particle to boundary and reflect
        mesh.distance_to_boundary(state);
        mesh.move_to_surface(state);
        reflected[tid] = mesh.reflect(state);

        dirs_out[tid] = state.d_dir;
        ijk_out[tid]  = state.ijk;
        exit_face[tid] = state.exiting_face;
        refl_face[tid] = state.reflecting_face;
    }
}

namespace
{

// Build Mesh_Geometry
std::shared_ptr<Mesh_Geometry_DMM> get_mesh()
{
    std::vector<double> x_edges = {0.0, 0.1, 0.6, 0.9, 1.0};
    std::vector<double> y_edges = {-1.0, -0.6, 0.0};
    std::vector<double> z_edges = {2.0, 2.6, 3.4, 4.0};
    
    auto mesh = std::make_shared<Mesh_Geometry_DMM>(x_edges,y_edges,z_edges);
    return mesh;
}

}

//---------------------------------------------------------------------------//
// Compute volumes of specified cells
//---------------------------------------------------------------------------//
void Mesh_Geometry_Tester::test_volume()
{
    auto mesh = get_mesh();

    std::vector<cell_type> cells = {4, 1, 22, 11};
    int num_cells = cells.size();

    const auto& all_volumes = mesh->volumes();
    std::vector<double> cell_volumes(num_cells);

    for (int cellid = 0; cellid < num_cells; ++cellid)
        cell_volumes[cellid] = all_volumes[cells[cellid]];

    std::vector<double> expected_volumes = {0.1 * 0.6 * 0.6,
                                            0.5 * 0.4 * 0.6,
                                            0.3 * 0.6 * 0.6,
                                            0.1 * 0.4 * 0.8};

    EXPECT_VEC_SOFT_EQ( expected_volumes, cell_volumes );
}

//---------------------------------------------------------------------------//
// Compute matids of specified cells
//---------------------------------------------------------------------------//
void Mesh_Geometry_Tester::test_matid()
{
    auto mesh = get_mesh();

    std::vector<int> all_matids = {1, 3, 2, 0,
                                   3, 1, 4, 1,
                                   2, 5, 2, 1,
                                   0, 1, 2, 3,
                                   1, 2, 3, 4,
                                   2, 3, 4, 5};

    std::vector<Point> host_points = {{0.7,  -0.9,  2.1},
                                      {0.5,  -0.5,  2.5},
                                      {0.99, -0.01, 3.99},
                                      {0.05, -0.8,  2.4}};

    int num_points = host_points.size();
        
    // Set matids with mesh (host call)
    mesh->set_matids(all_matids);

    // Create memory on device
    thrust::device_vector<Point> device_points(host_points);
    thrust::device_vector<int> device_cell_matids(num_points);

    // Execute kernel
    compute_matids_kernel<<<1,num_points>>>(
             mesh->device_instance(),
             num_points,
             device_points.data().get(),
             device_cell_matids.data().get());

    REQUIRE( hipGetLastError() == hipSuccess );

    // Copy matids back to host
    std::vector<int> host_cell_matids(num_points);
    thrust::copy(device_cell_matids.begin(),device_cell_matids.end(),
                 host_cell_matids.begin());

    std::vector<int> expected_matids = {2, 1, 5, 1};

    EXPECT_VEC_EQ( expected_matids, host_cell_matids);
}

//---------------------------------------------------------------------------//
// Test distance to boundary
//---------------------------------------------------------------------------//
void Mesh_Geometry_Tester::test_dist_to_bdry()
{
    using def::I;  using def::J; using def::K;

    // Build mesh
    auto x_edges = {0.0, 0.10, 0.25, 0.30, 0.42};
    auto y_edges = {0.0, 0.20, 0.40, 0.50};
    auto z_edges = {-0.1, 0.0, 0.15, 0.50};
    auto mesh = std::make_shared<Mesh_Geometry_DMM>(x_edges,y_edges,z_edges);


    double sqrt_half = sqrt(0.5);
    std::vector<Point> host_points = {{0.01, 0.01, -0.01},
                                      {0.26, 0.35, -0.01}};
    std::vector<Point> host_dirs   = {{1.0,  0.0,   0.0},
                                      {sqrt_half, sqrt_half, 0.0}};

    int num_points = host_points.size();

    // Create memory on device
    thrust::device_vector<Point> device_points(host_points);
    thrust::device_vector<Point> device_dirs(host_dirs);
    thrust::device_vector<double> device_distances(num_points);
    thrust::device_vector<Coords> device_coords(num_points);

    // Execute kernel
    distance_kernel<<<1,num_points>>>(
             mesh->device_instance(),
             num_points,
             device_points.data().get(),
             device_dirs.data().get(),
             device_distances.data().get(),
             device_coords.data().get());

    REQUIRE( hipGetLastError() == hipSuccess );

    // Copy data back to host
    std::vector<double> host_distances(num_points);
    thrust::copy(device_distances.begin(),device_distances.end(),
                 host_distances.begin());
    std::vector<Coords> host_coords(num_points);
    thrust::copy(device_coords.begin(),device_coords.end(),
                 host_coords.begin());

    std::vector<double> expected_distances = {0.1 - 0.01, 0.04 / sqrt_half};
    std::vector<Coords> expected_coords = {{1, 0, 0}, {3, 1, 0}};

    EXPECT_VEC_SOFT_EQ(expected_distances, host_distances);
    EXPECT_EQ(expected_coords[0][I], host_coords[0][I]);
    EXPECT_EQ(expected_coords[0][J], host_coords[0][J]);
    EXPECT_EQ(expected_coords[0][K], host_coords[0][K]);
    EXPECT_EQ(expected_coords[1][I], host_coords[1][I]);
    EXPECT_EQ(expected_coords[1][J], host_coords[1][J]);
    EXPECT_EQ(expected_coords[1][K], host_coords[1][K]);
}

//---------------------------------------------------------------------------//
// Test move to surface
//---------------------------------------------------------------------------//
void Mesh_Geometry_Tester::test_move_to_surf()
{
    using def::I;  using def::J; using def::K;

    // Build mesh
    auto x_edges = {0.0, 0.10, 0.25, 0.30, 0.42};
    auto y_edges = {0.0, 0.20, 0.40, 0.50};
    auto z_edges = {-0.1, 0.0, 0.15, 0.50};
    auto mesh = std::make_shared<Mesh_Geometry_DMM>(x_edges,y_edges,z_edges);

    double sqrt_half = sqrt(0.5);
    std::vector<Point> host_points = {{0.01, 0.01, -0.01},
                                      {0.26, 0.35, -0.01}};
    std::vector<Point> host_dirs   = {{1.0,  0.0,   0.0},
                                      {sqrt_half, sqrt_half, 0.0}};

    int num_points = host_points.size();

    // Create memory on device
    thrust::device_vector<Point> device_points(host_points);
    thrust::device_vector<Point> device_dirs(host_dirs);
    thrust::device_vector<Coords> device_coords(num_points);

    // Execute kernel
    move_to_surf_kernel<<<1,num_points>>>(
             mesh->device_instance(),
             num_points,
             device_points.data().get(),
             device_dirs.data().get(),
             device_coords.data().get());

    REQUIRE( hipGetLastError() == hipSuccess );

    // Copy data back to host
    std::vector<Coords> host_coords(num_points);
    thrust::copy(device_coords.begin(),device_coords.end(),
                 host_coords.begin());

    std::vector<Coords> expected_coords = {{1, 0, 0}, {3, 1, 0}};

    EXPECT_EQ(expected_coords[0][I], host_coords[0][I]);
    EXPECT_EQ(expected_coords[0][J], host_coords[0][J]);
    EXPECT_EQ(expected_coords[0][K], host_coords[0][K]);
    EXPECT_EQ(expected_coords[1][I], host_coords[1][I]);
    EXPECT_EQ(expected_coords[1][J], host_coords[1][J]);
    EXPECT_EQ(expected_coords[1][K], host_coords[1][K]);
}

//---------------------------------------------------------------------------//
// Test reflection
//---------------------------------------------------------------------------//
void Mesh_Geometry_Tester::test_reflect()
{
    using def::I;  using def::J; using def::K;
    typedef cuda_profugus::Mesh_State Geo_State_t;

    // Build mesh
    auto x_edges = {0.0, 0.10, 0.25, 0.30, 0.42};
    auto y_edges = {0.0, 0.20, 0.40, 0.50};
    auto z_edges = {-0.1, 0.0, 0.15, 0.50};
    auto mesh = std::make_shared<Mesh_Geometry_DMM>(x_edges,y_edges,z_edges);

    std::vector<int> refl = {1, 0, 0, 0, 1, 1};
    mesh->set_reflecting(refl);

    std::vector<Point> host_points = {{0.05, 0.42, 0.10},
                                      {0.20, 0.45, 0.35}};
    std::vector<Point> host_dirs   = {{-4.0, 0.1, -0.5},
                                      {-1.0, 2.0, 1.0}};
    // Normalize directions
    double nrm = std::sqrt(host_dirs[0][I]*host_dirs[0][I] +
                           host_dirs[0][J]*host_dirs[0][J] +
                           host_dirs[0][K]*host_dirs[0][K]);
    host_dirs[0][I] /= nrm;
    host_dirs[0][J] /= nrm;
    host_dirs[0][K] /= nrm;

    nrm = std::sqrt(host_dirs[1][I]*host_dirs[1][I] +
                    host_dirs[1][J]*host_dirs[1][J] +
                    host_dirs[1][K]*host_dirs[1][K]);
    host_dirs[1][I] /= nrm;
    host_dirs[1][J] /= nrm;
    host_dirs[1][K] /= nrm;

    int num_points = host_points.size();

    // Create memory on device
    thrust::device_vector<Point> device_points(host_points);
    thrust::device_vector<Point> device_dirs_in(host_dirs);
    thrust::device_vector<Point>  device_dirs_out(num_points);
    thrust::device_vector<Coords> device_ijk_out(num_points);
    thrust::device_vector<int> device_exit_face(num_points);
    thrust::device_vector<int> device_refl_face(num_points);
    thrust::device_vector<int> device_reflected(num_points);

    // Execute kernel
    reflect_kernel<<<1,num_points>>>(
             mesh->device_instance(),
             num_points,
             device_points.data().get(),
             device_dirs_in.data().get(),
             device_reflected.data().get(),
             device_dirs_out.data().get(),
             device_ijk_out.data().get(),
             device_exit_face.data().get(),
             device_refl_face.data().get());

    REQUIRE( hipGetLastError() == hipSuccess );

    // Test reflected directions
    std::vector<Point> host_dirs_out(num_points);
    thrust::copy(device_dirs_out.begin(),device_dirs_out.end(),
                 host_dirs_out.begin());
    std::vector<Point> expected_dirs = {{-host_dirs[0][I],
                                         host_dirs[0][J],
                                         host_dirs[0][K]},
                                        {host_dirs[1][I],
                                         host_dirs[1][J],
                                         host_dirs[1][K]}};
    EXPECT_SOFT_EQ(expected_dirs[0][I], host_dirs_out[0][I]);
    EXPECT_SOFT_EQ(expected_dirs[0][J], host_dirs_out[0][J]);
    EXPECT_SOFT_EQ(expected_dirs[0][K], host_dirs_out[0][K]);
    EXPECT_SOFT_EQ(expected_dirs[1][I], host_dirs_out[1][I]);
    EXPECT_SOFT_EQ(expected_dirs[1][J], host_dirs_out[1][J]);
    EXPECT_SOFT_EQ(expected_dirs[1][K], host_dirs_out[1][K]);

    // Test reflected cell indices
    std::vector<Coords> host_ijk_out(num_points);
    thrust::copy(device_ijk_out.begin(),device_ijk_out.end(),
                 host_ijk_out.begin());
    std::vector<Coords> expected_ijk = {{0, 2, 1}, {1, 3, 2}};
    EXPECT_EQ(expected_ijk[0][I], host_ijk_out[0][I]);
    EXPECT_EQ(expected_ijk[0][J], host_ijk_out[0][J]);
    EXPECT_EQ(expected_ijk[0][K], host_ijk_out[0][K]);
    EXPECT_EQ(expected_ijk[1][I], host_ijk_out[1][I]);
    EXPECT_EQ(expected_ijk[1][J], host_ijk_out[1][J]);
    EXPECT_EQ(expected_ijk[1][K], host_ijk_out[1][K]);

    // Test reflected flag
    std::vector<int> host_reflected(num_points);
    thrust::copy(device_reflected.begin(),device_reflected.end(),
                 host_reflected.begin());
    std::vector<int> expected_reflected= {1, 0};
    EXPECT_VEC_EQ(expected_reflected, host_reflected);

    // Test exiting faces
    std::vector<int> host_exit_face(num_points);
    thrust::copy(device_exit_face.begin(),device_exit_face.end(),
                 host_exit_face.begin());
    std::vector<int> expected_exit_face = {Geo_State_t::MINUS_X,
                                           Geo_State_t::PLUS_Y};
    EXPECT_VEC_EQ(expected_exit_face, host_exit_face);

    // Test reflecting faces
    std::vector<int> host_refl_face(num_points);
    thrust::copy(device_refl_face.begin(),device_refl_face.end(),
                 host_refl_face.begin());
    std::vector<int> expected_refl_face = {Geo_State_t::MINUS_X,
                                           Geo_State_t::NONE};
    EXPECT_VEC_EQ(expected_refl_face, host_refl_face);
}

//---------------------------------------------------------------------------//
// Test reflection
//---------------------------------------------------------------------------//
void Mesh_Geometry_Tester::test_reflect()
{
    typedef cuda_profugus::Mesh_State Geo_State_t;

    // Build mesh
    auto x_edges = {0.0, 0.10, 0.25, 0.30, 0.42};
    auto y_edges = {0.0, 0.20, 0.40, 0.50};
    auto z_edges = {-0.1, 0.0, 0.15, 0.50};
    auto mesh = std::make_shared<Mesh_Geometry>(x_edges,y_edges,z_edges);

    std::vector<int> refl = {1, 0, 0, 0, 1, 1};
    mesh->set_reflecting(refl);

    std::vector<Point> host_points = {{0.05, 0.42, 0.10},
                                      {0.20, 0.45, 0.35}};
    std::vector<Point> host_dirs   = {{-4.0, 0.1, -0.5},
                                      {-1.0, 2.0, 1.0}};
    // Normalize directions
    double nrm = std::sqrt(host_dirs[0].x*host_dirs[0].x +
                           host_dirs[0].y*host_dirs[0].y +
                           host_dirs[0].z*host_dirs[0].z);
    host_dirs[0].x /= nrm;
    host_dirs[0].y /= nrm;
    host_dirs[0].z /= nrm;

    nrm = std::sqrt(host_dirs[1].x*host_dirs[1].x +
                    host_dirs[1].y*host_dirs[1].y +
                    host_dirs[1].z*host_dirs[1].z);
    host_dirs[1].x /= nrm;
    host_dirs[1].y /= nrm;
    host_dirs[1].z /= nrm;

    int num_points = host_points.size();

    // Create memory on device
    thrust::device_vector<Point> device_points(host_points);
    thrust::device_vector<Point> device_dirs_in(host_dirs);
    thrust::device_vector<Point>  device_dirs_out(num_points);
    thrust::device_vector<Coords> device_ijk_out(num_points);
    thrust::device_vector<int> device_exit_face(num_points);
    thrust::device_vector<int> device_refl_face(num_points);
    thrust::device_vector<int> device_reflected(num_points);

    // Execute kernel
    reflect_kernel<<<1,num_points>>>(
            *mesh,
             num_points,
             device_points.data().get(),
             device_dirs_in.data().get(),
             device_reflected.data().get(),
             device_dirs_out.data().get(),
             device_ijk_out.data().get(),
             device_exit_face.data().get(),
             device_refl_face.data().get());

    REQUIRE( hipGetLastError() == hipSuccess );

    // Test reflected directions
    std::vector<Point> host_dirs_out(num_points);
    thrust::copy(device_dirs_out.begin(),device_dirs_out.end(),
                 host_dirs_out.begin());
    std::vector<Point> expected_dirs = {{-host_dirs[0].x,
                                         host_dirs[0].y,
                                         host_dirs[0].z},
                                        {host_dirs[1].x,
                                         host_dirs[1].y,
                                         host_dirs[1].z}};
    EXPECT_SOFT_EQ(expected_dirs[0].x, host_dirs_out[0].x);
    EXPECT_SOFT_EQ(expected_dirs[0].y, host_dirs_out[0].y);
    EXPECT_SOFT_EQ(expected_dirs[0].z, host_dirs_out[0].z);
    EXPECT_SOFT_EQ(expected_dirs[1].x, host_dirs_out[1].x);
    EXPECT_SOFT_EQ(expected_dirs[1].y, host_dirs_out[1].y);
    EXPECT_SOFT_EQ(expected_dirs[1].z, host_dirs_out[1].z);

    // Test reflected cell indices
    std::vector<Coords> host_ijk_out(num_points);
    thrust::copy(device_ijk_out.begin(),device_ijk_out.end(),
                 host_ijk_out.begin());
    std::vector<Coords> expected_ijk = {{0, 2, 1}, {1, 3, 2}};
    EXPECT_EQ(expected_ijk[0].i, host_ijk_out[0].i);
    EXPECT_EQ(expected_ijk[0].j, host_ijk_out[0].j);
    EXPECT_EQ(expected_ijk[0].k, host_ijk_out[0].k);
    EXPECT_EQ(expected_ijk[1].i, host_ijk_out[1].i);
    EXPECT_EQ(expected_ijk[1].j, host_ijk_out[1].j);
    EXPECT_EQ(expected_ijk[1].k, host_ijk_out[1].k);

    // Test reflected flag
    std::vector<int> host_reflected(num_points);
    thrust::copy(device_reflected.begin(),device_reflected.end(),
                 host_reflected.begin());
    std::vector<int> expected_reflected= {1, 0};
    EXPECT_VEC_EQ(expected_reflected, host_reflected);

    // Test exiting faces
    std::vector<int> host_exit_face(num_points);
    thrust::copy(device_exit_face.begin(),device_exit_face.end(),
                 host_exit_face.begin());
    std::vector<int> expected_exit_face = {Geo_State_t::MINUS_X,
                                           Geo_State_t::PLUS_Y};
    EXPECT_VEC_EQ(expected_exit_face, host_exit_face);

    // Test reflecting faces
    std::vector<int> host_refl_face(num_points);
    thrust::copy(device_refl_face.begin(),device_refl_face.end(),
                 host_refl_face.begin());
    std::vector<int> expected_refl_face = {Geo_State_t::MINUS_X,
                                           Geo_State_t::NONE};
    EXPECT_VEC_EQ(expected_refl_face, host_refl_face);
}

//---------------------------------------------------------------------------//
// end of MC/cuda_geometry/test/Mesh_Geometry_Tester.cu
//---------------------------------------------------------------------------//
