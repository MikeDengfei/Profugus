#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
/*!
 * \file   cuda_mc/test/Tally_Tester.cu
 * \author Stuart Slattery
 * \note   Copyright (C) 2013 Oak Ridge National Laboratory, UT-Battelle, LLC.
 */
//---------------------------------------------------------------------------//

#include "Tally_Tester.hh"

#include "cuda_utils/Hardware.hh"
#include "cuda_utils/CudaDBC.hh"

#include <Teuchos_Array.hpp>

#include <hip/hip_runtime.h>

//---------------------------------------------------------------------------//
// CUDA Kernels
//---------------------------------------------------------------------------//
__global__ 
void set_wt_kernel( Tally_Tester::Particle_Vector* vector, 
		    double* wt )
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    vector->set_wt( i, wt[i] );
    vector->set_step( i, 2.0 );
}

//---------------------------------------------------------------------------//
__global__ 
void set_event_kernel( Tally_Tester::Particle_Vector* vector, 
		       typename Tally_Tester::Event_t* event )
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    vector->set_event( i, event[i] );
}

//---------------------------------------------------------------------------//
__global__ 
void set_geo_state_kernel( 
    Tally_Tester::Particle_Vector* vector, 
    typename Tally_Tester::Geo_State_t* geo_state )
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    vector->geo_state( i ) = geo_state[i];
}

//---------------------------------------------------------------------------//
__global__ 
void set_batch_kernel( Tally_Tester::Particle_Vector* vector, 
		       int* batch )
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    vector->set_batch( i, batch[i] );
}

//---------------------------------------------------------------------------//
__global__ 
void live_kernel( Tally_Tester::Particle_Vector* vector )
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    vector->live( i );
}

//---------------------------------------------------------------------------//
// Tally_Tester
//---------------------------------------------------------------------------//
Tally_Tester::Tally_Tester(  const std::vector<double>& x_edges,
						 const std::vector<double>& y_edges,
						 const std::vector<double>& z_edges,
						 const int num_particle, 
						 const profugus::RNG& rng,
						 const int num_batch )
{
    // Acquire hardware for the test.
    cuda_utils::Hardware<cuda_utils::arch::Device>::acquire();

    // Create the geometry.
    auto geom_dmm = std::make_shared<Geometry_DMM>(x_edges,y_edges,z_edges);
    d_geometry = cuda_utils::shared_device_ptr<Geometry>(geom_dmm->device_instance());
    int num_cells = geom_dmm->num_cells();

    // Create the particle vector.
    d_particles = cuda_utils::shared_device_ptr<Particle_Vector>( num_particle, rng );

    // Initialize the state of the particle vector. The first half of the
    // cells will have a particle with a collision event, the rest will
    // not. There will be one particle per cell per batch.

    // Cuda launch params.
    int num_threads = 256;
    int num_block = num_particle / num_threads;

    // Set the weight. Weight assigned as a function of the cell id and the
    // batch id. This will also set all particles to a step length of 2.
    Teuchos::Array<double> wt( num_particle );
    for ( int b = 0; b < num_batch; ++b )
    {
	for ( int c = 0; c < num_cells; ++c )
	{
	    wt[ b*num_cells + c ] = (b+1)*(c+1);
	}
    }
    double* device_wt;
    hipMalloc( (void**) &device_wt, num_particle * sizeof(double) );
    hipMemcpy( device_wt, wt.getRawPtr(), num_particle * sizeof(double),
		hipMemcpyHostToDevice );
    set_wt_kernel<<<num_block,num_threads>>>( 
	d_particles.get_device_ptr(), device_wt );
    hipFree( device_wt );

    // Set the event. If the particle is not in the first cells it does not
    // collide.
    Teuchos::Array<Event_t> events( num_particle );
    for ( int b = 0; b < num_batch; ++b )
    {
	for ( int c = 0; c < num_cells; ++c )
	{
	    if ( c < num_cells / 2 )
	    {
		events[ b*num_cells + c ] = cuda_profugus::events::COLLISION;
	    }
	    else
	    {
		events[ b*num_cells + c ] = cuda_profugus::events::ABSORPTION;
	    }
	}
    }
    Event_t* device_event;
    hipMalloc( (void**) &device_event, num_particle * sizeof(Event_t) );
    hipMemcpy( device_event, events.getRawPtr(), num_particle * sizeof(Event_t),
		hipMemcpyHostToDevice );
    set_event_kernel<<<num_block,num_threads>>>( 
	d_particles.get_device_ptr(), device_event );
    hipFree( device_event );

    // Set the batch
    Teuchos::Array<int> batch( num_particle );
    for ( int b = 0; b < num_batch; ++b )
    {
	for ( int c = 0; c < num_cells; ++c )
	{
	    batch[ b*num_cells + c ] = b;
	}
    }
    int* device_batch;
    hipMalloc( (void**) &device_batch, num_particle * sizeof(int) );
    hipMemcpy( device_batch, batch.getRawPtr(), num_particle * sizeof(int),
		hipMemcpyHostToDevice );
    set_batch_kernel<<<num_block,num_threads>>>( 
	d_particles.get_device_ptr(), device_batch );
    hipFree( device_batch );

    // Set the geometry state. One particle per cell per batch.
    Teuchos::Array<Geo_State_t> geo_state( num_particle );
    int cells_x = x_edges.size()-1;
    int cells_y = y_edges.size()-1;
    int cells_z = z_edges.size()-1;
    for ( int b = 0; b < num_batch; ++b )
    {
        for (int k = 0; k < cells_z; ++k)
        {
            for (int j = 0; j < cells_y; ++j)
            {
                for (int i = 0; i < cells_x; ++i)
                {
                    int c = i + cells_x * (j + cells_y * k);
                    geo_state[ b*num_cells + c ].ijk[0] = i;
                    geo_state[ b*num_cells + c ].ijk[1] = j;
                    geo_state[ b*num_cells + c ].ijk[2] = k;
                }
            }
        }
    }
    Geo_State_t* device_geo_state;
    hipMalloc( (void**) &device_geo_state, num_particle * sizeof(Geo_State_t) );
    hipMemcpy( device_geo_state, geo_state.getRawPtr(), num_particle * sizeof(Geo_State_t),
		hipMemcpyHostToDevice );
    set_geo_state_kernel<<<num_block,num_threads>>>( 
	d_particles.get_device_ptr(), device_geo_state );
    hipFree( device_geo_state );

    // Set the particles to alive.
    live_kernel<<<num_block,num_threads>>>( d_particles.get_device_ptr() );

    // sort the particles by event.
    d_particles.get_host_ptr()->sort_by_event( d_particles.get_host_ptr()->size() );
}

//---------------------------------------------------------------------------//
//                 end of cuda_mc/Tally_Tester.cu
//---------------------------------------------------------------------------//
