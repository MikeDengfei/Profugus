#include "hip/hip_runtime.h"
//----------------------------------*-C++-*----------------------------------//
/*!
 * \file   cuda_mc/Source_Transporter_Tester.cc
 * \author Steven Hamilton
 * \date   Wed Jan 20 16:13:24 2016
 * \brief  Source_Transporter_Tester member definitions.
 * \note   Copyright (C) 2016 Oak Ridge National Laboratory, UT-Battelle, LLC.
 */
//---------------------------------------------------------------------------//

#include "Source_Transporter_Tester.hh"
#include "Test_XS.hh"
#include "../Source_Transporter.hh"
#include "../Uniform_Source.cuh"
#include "../Physics.cuh"
#include "../Cell_Tally.cuh"
#include "../Tallier.cuh"
#include "gtest/Gtest_Functions.hh"
#include "cuda_geometry/Mesh_Geometry.hh"
#include "CudaUtils/cuda_utils/Shared_Device_Ptr.hh"
#include "Teuchos_RCP.hpp"
#include "Teuchos_ParameterList.hpp"

using namespace cuda_mc;

typedef cuda_profugus::Mesh_Geometry      Geom;
typedef cuda_profugus::Mesh_Geometry_DMM  Geom_DMM;
typedef cuda_mc::Uniform_Source<Geom>     Uniform_Src;
typedef cuda_mc::Uniform_Source_DMM<Geom> Uniform_Src_DMM;
typedef cuda_mc::Source_Transporter<Geom> Transporter;

void Source_Transporter_Tester::test_transport(int num_groups)
{
    auto xs = Test_XS::build_xs(num_groups);

    def::size_type Np = 10000;

    // Build geometry
    std::vector<double> edges = {0.0, 0.50, 1.0};
    std::vector<int> matids;
    if (num_groups == 1)
        matids = {0, 0, 0, 0, 0, 0, 0, 0};
    else
        matids = {0, 1, 1, 0, 0, 1, 1, 0};
    auto geom_dmm = std::make_shared<Geom_DMM>(edges,edges,edges);
    geom_dmm->set_matids(matids);
    auto sdp_geom = cuda::shared_device_ptr<Geom>(geom_dmm->device_instance());

    // Build physics
    Teuchos::RCP<Teuchos::ParameterList> pl( new Teuchos::ParameterList() );
    pl->set("num_groups",xs->num_groups());
    pl->set("Np",Np);
    pl->set("implicit_capture",true);
    pl->set("variance reduction",std::string("roulette"));
    auto sdp_mat = cuda::shared_device_ptr<cuda_profugus::XS_Device>(*xs);
    auto phys = std::make_shared<Physics<Geom> >(pl,xs,sdp_mat);
    phys->set_geometry(sdp_geom);
    cuda::Shared_Device_Ptr<Physics<Geom> > sdp_phys(phys);

    // Build cell tally
    std::cout << "Building Cell_Tally" << std::endl;
    auto sp_cell_tally = std::make_shared<Cell_Tally<Geom>>(
        sdp_geom,sdp_phys);
    std::vector<int> cells = {0, 1, 2, 3, 4, 5, 6, 7};
    sp_cell_tally->set_cells(cells,geom_dmm->volumes());
    cuda::Shared_Device_Ptr<Cell_Tally<Geom> > cell_tally(sp_cell_tally);

    std::cout << "Building Tallier" << std::endl;
    auto sp_tallier = std::make_shared<Tallier<Geom> >();
    sp_tallier->add_cell_tally(cell_tally);
    cuda::Shared_Device_Ptr<Tallier<Geom>> tallier(sp_tallier);

    // Build box shape for source
    std::vector<double> src_bounds = {edges.front(), edges.back(),
                                      edges.front(), edges.back(),
                                      edges.front(), edges.back()};
    REQUIRE( src_bounds.size() == 6 );
    auto src_shape = cuda::shared_device_ptr<cuda_mc::Box_Shape>(
            src_bounds[0], src_bounds[1],
            src_bounds[2], src_bounds[3],
            src_bounds[4], src_bounds[5]);

    // Build source
    auto source_dmm = std::make_shared<Uniform_Src_DMM>(pl,sdp_geom);
    source_dmm->build_source(src_shape);

    // Build source transporter
    pl->set("batch_size",1000);
    pl->set("sort_frequency",4);
    pl->set("verbosity",std::string("high"));
    Transporter trans(pl,sdp_geom,sdp_phys);
    trans.set(tallier);
    trans.solve(source_dmm);

    sp_tallier->finalize(Np);
    auto tally = sp_cell_tally->results();
    std::cout << "Tally result: ";
    for( auto x : tally )
        std::cout << x << " ";
    std::cout << std::endl;

    // Test statistics on output using symmetry
    EXPECT_EQ( tally.size(), 8 );

    if (num_groups==1)
    {
        double mean = 0.0;
        for( auto x : tally )
            mean += x;
        mean /= static_cast<double>(tally.size());

        double tol = 10.0 / std::sqrt( static_cast<double>(Np) );

        std::vector<double> exp(8,mean);
        EXPECT_VEC_SOFTEQ( exp, tally, tol );
    }
    else
    {
        double mean0 = 0.0;
        double mean1 = 0.0;
        int count0 = 0;
        int count1 = 0;
        for( int cell = 0; cell < tally.size(); ++cell )
        {
            if( matids[cell] == 0 )
            {
                mean0 += tally[cell];
                count0++;
            }
            else if( matids[cell] == 1 )
            {
                mean1 += tally[cell];
                count1++;
            }
        }
        mean0 /= static_cast<double>(count0);
        mean1 /= static_cast<double>(count1);

        double tol = 10.0 / std::sqrt( static_cast<double>(Np) );

        std::vector<double> exp(8);
        for( int cell = 0; cell < matids.size(); ++cell )
        {
            if( matids[cell] == 0 )
                exp[cell] = mean0;
            else if( matids[cell] == 1 )
                exp[cell] = mean1;
        }
        EXPECT_VEC_SOFTEQ( exp, tally, tol );
    }
}

//---------------------------------------------------------------------------//
//                 end of Source_Transporter_Tester.cc

//---------------------------------------------------------------------------//
