#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
/*!
 * \file   cuda_mc/test/Physics_Tester.cu
 * \author Stuart Slattery
 * \note   Copyright (C) 2013 Oak Ridge National Laboratory, UT-Battelle, LLC.
 */
//---------------------------------------------------------------------------//

#include "Physics_Tester.hh"

#include "cuda_utils/Hardware.hh"
#include "cuda_utils/CudaDBC.hh"
#include "cuda_utils/Utility_Functions.hh"
#include "cuda_utils/Memory.cuh"

#include <Teuchos_Array.hpp>

#include <hip/hip_runtime.h>

//---------------------------------------------------------------------------//
// CUDA Kernels
//---------------------------------------------------------------------------//
__global__ void geometry_initialize_kernel( 
    Physics_Tester::Particle_Vector* vector, 
    const Physics_Tester::Geometry* geometry,
    const Physics_Tester::Space_Vector r,
    const Physics_Tester::Space_Vector d,
    const int matid,
    const int num_particle )
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if ( i < num_particle ) 
    {
	geometry->initialize( r, d, vector->geo_state(i) );
	vector->set_matid( i, matid );
	vector->set_event( i, cuda_profugus::events::COLLISION );
	vector->set_wt( i, 0.9 );
        vector->set_dist_mfp( i, 500.0 );
    }
}

//---------------------------------------------------------------------------//
__global__ void sample_group_kernel( 
    Physics_Tester::Particle_Vector* vector,
    const double* cdf,
    const int cdf_size,
    const int num_particle )
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if ( i < num_particle ) 
    {
	int group =
	    cuda_utils::utility::sample_discrete_CDF( cdf_size, cdf, vector->ran(i) );
	vector->set_group( i, group );
    }
}

//---------------------------------------------------------------------------//
__global__ void is_fissionable_kernel( const Physics_Tester::Physics* physics,
				       const int matid,
				       int* is_f )
{
    *is_f = physics->is_fissionable( matid );
}

//---------------------------------------------------------------------------//
__global__ void total_kernel( const Physics_Tester::Physics* physics,
			      const cuda_profugus::physics::Reaction_Type type,
			      const int matid,
			      const int group,
			      double* total )
{
    *total = physics->total( type, matid, group );
}

//---------------------------------------------------------------------------//
__global__ void min_max_energy_kernel( const Physics_Tester::Physics* physics,
				       double* min_max_energy )
{
    min_max_energy[0] = physics->min_energy();
    min_max_energy[1] = physics->max_energy();
}

//---------------------------------------------------------------------------//
__global__ void initialize_fission_from_spectrum_kernel( 
    const Physics_Tester::Physics* physics,
    const int matid,
    const double ran,
    int* result )					   
{
    bool sampled = false;
    physics->initialize_fission( matid, ran, result[0], sampled);
    result[1] = sampled;
}

//---------------------------------------------------------------------------//
__global__ void initialize_fission_from_site_kernel( 
    const Physics_Tester::Physics* physics,
    const Physics_Tester::Fission_Site fs,
    const double ran,
    int* result )
{
    bool sampled = false;
    physics->initialize_fission( fs, ran, result[0], sampled );
    result[1] = sampled;
}

//---------------------------------------------------------------------------//
// Physics_Tester
//---------------------------------------------------------------------------//
Physics_Tester::Physics_Tester( 
    const std::vector<double>& x_edges,
    const std::vector<double>& y_edges,
    const std::vector<double>& z_edges,
    const int vector_size,
    const profugus::RNG& rng,
    Teuchos::ParameterList& db,
    const profugus::XS& xs,
    const int matid )
    : d_size( vector_size )
    , d_particle_tester( vector_size, rng )
{
    // Acquire hardware for the test.
    if ( !cuda_utils::Hardware<cuda_utils::arch::Device>::have_acquired() )
	cuda_utils::Hardware<cuda_utils::arch::Device>::acquire();

    // Create the geometry on the host.
    d_host_geom = std::make_shared<Geometry_DMM>( x_edges, y_edges, z_edges );
    int num_cells = d_host_geom->num_cells();

    // Set matids with the geometry on the host.
    std::vector<int> matids( num_cells, matid );
    d_host_geom->set_matids( matids );

    // Create a device copy of the geometry.
    d_geometry = cuda_utils::shared_device_ptr<Geometry>(
        d_host_geom->device_instance());

    // Create the physics.
    d_physics = cuda_utils::shared_device_ptr<Physics>( db, xs );

    // Set the geometry with the physics.
    d_physics.get_host_ptr()->set_geometry( d_geometry );

    // Create the source shape.
    d_shape = cuda_utils::shared_device_ptr<Shape>( 
	*std::min_element(x_edges.begin(),x_edges.end()),
	*std::max_element(x_edges.begin(),x_edges.end()),
	*std::min_element(y_edges.begin(),y_edges.end()),
	*std::max_element(y_edges.begin(),y_edges.end()),
	*std::min_element(z_edges.begin(),z_edges.end()),
	*std::max_element(z_edges.begin(),z_edges.end()) );
}

//---------------------------------------------------------------------------//
// Initialize particles with the geometry and set to collide.
void Physics_Tester::geometry_initialize( 
    const Space_Vector r, const Space_Vector d, const int matid )
{
    unsigned int threads_per_block = 
	cuda_utils::Hardware<cuda_utils::arch::Device>::default_block_size();
    unsigned int num_blocks = d_size / threads_per_block;
    if ( d_size % threads_per_block > 0 ) ++num_blocks;

    geometry_initialize_kernel<<<num_blocks,threads_per_block>>>(
        particles().get_device_ptr(), d_geometry.get_device_ptr(),
        r, d, matid, d_size );

    particles().get_host_ptr()->sort_by_event( particles().get_host_ptr()->size() );
}

//---------------------------------------------------------------------------//
// Sample a cdf and set the particle group.
void Physics_Tester::sample_group( const std::vector<double>& cdf )
{
    // copy the cdf to the device.
    double* device_cdf;
    cuda_utils::memory::Malloc( device_cdf, cdf.size() );
    cuda_utils::memory::Copy_To_Device( device_cdf, cdf.data(), cdf.size() );

    // Sample the cdf and set the particle groups.
    unsigned int threads_per_block = 
	cuda_utils::Hardware<cuda_utils::arch::Device>::default_block_size();
    unsigned int num_blocks = d_size / threads_per_block;
    if ( d_size % threads_per_block > 0 ) ++num_blocks;

    sample_group_kernel<<<num_blocks,threads_per_block>>>(
	particles().get_device_ptr(), device_cdf, cdf.size(), d_size );

    // free allocated data.
    cuda_utils::memory::Free( device_cdf );
}

//---------------------------------------------------------------------------//
// Check if a matid is fissionable.
bool Physics_Tester::is_fissionable(const int matid) const
{
    int* is_f_device;
    cuda_utils::memory::Malloc( is_f_device, 1 );
    is_fissionable_kernel<<<1,1>>>( d_physics.get_device_ptr(), matid, is_f_device );

    int is_f_host = 0;
    cuda_utils::memory::Copy_To_Host( &is_f_host, is_f_device, 1 );
    cuda_utils::memory::Free( is_f_device );
    return is_f_host;
}

//---------------------------------------------------------------------------//
// get a total cross section
double Physics_Tester::get_total(
    const int matid,
    const int group,
    const cuda_profugus::physics::Reaction_Type type ) const
{
    double* total_device;
    cuda_utils::memory::Malloc( total_device, 1 );
    total_kernel<<<1,1>>>( d_physics.get_device_ptr(),
			   type,
			   matid,
			   group,
			   total_device );

    double total_host = 0;
    cuda_utils::memory::Copy_To_Host( &total_host, total_device, 1 );
    cuda_utils::memory::Free( total_device );
    return total_host;
}

//---------------------------------------------------------------------------//
// get the min and max particle energies
void Physics_Tester::get_min_max_energy( double& min, double& max ) const
{
    double* minmax_device;
    cuda_utils::memory::Malloc( minmax_device, 2 );
    min_max_energy_kernel<<<1,1>>>( d_physics.get_device_ptr(),
				    minmax_device );

    double minmax_host[2];
    cuda_utils::memory::Copy_To_Host( minmax_host, minmax_device, 2 );
    cuda_utils::memory::Free( minmax_device );

    min = minmax_host[0];
    max = minmax_host[1];
}

//---------------------------------------------------------------------------//
// Initialize a particle from a fission specturm.
void Physics_Tester::initialize_fission_from_spectrum( const int matid,
						       const double ran,
						       int& group,
						       bool& sampled ) const
{
    int* result;
    cuda_utils::memory::Malloc( result, 2 );

    initialize_fission_from_spectrum_kernel<<<1,1>>>( d_physics.get_device_ptr(),
						      matid,
						      ran,
						      result );

    cuda_utils::memory::Copy_To_Host( &group, &result[0], 1 );
    int sampled_int = 0;
    cuda_utils::memory::Copy_To_Host( &sampled_int, &result[1], 1 );
    sampled = sampled_int;

    cuda_utils::memory::Free( result );
}

//---------------------------------------------------------------------------//
// Initialize a particle from a fission site.
void Physics_Tester::initialize_fission_from_site( const Fission_Site &fs,
						   const double ran,
						   int& group,
						   bool& sampled ) const
{
    int* result;
    cuda_utils::memory::Malloc( result, 2 );

    initialize_fission_from_site_kernel<<<1,1>>>( d_physics.get_device_ptr(),
						  fs,
						  ran,
						  result );

    cuda_utils::memory::Copy_To_Host( &group, &result[0], 1 );
    int sampled_int = 0;
    cuda_utils::memory::Copy_To_Host( &sampled_int, &result[1], 1 );
    sampled = sampled_int;

    cuda_utils::memory::Free( result );
}

//---------------------------------------------------------------------------//
//                 end of cuda_mc/Physics_Tester.cu
//---------------------------------------------------------------------------//
