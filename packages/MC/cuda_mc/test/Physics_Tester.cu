#include "hip/hip_runtime.h"
//----------------------------------*-C++-*----------------------------------//
/*!
 * \file   cuda_mc/Physics_Tester.cc
 * \author Steven Hamilton
 * \date   Wed Jan 20 16:13:24 2016
 * \brief  Physics_Tester member definitions.
 * \note   Copyright (C) 2016 Oak Ridge National Laboratory, UT-Battelle, LLC.
 */
//---------------------------------------------------------------------------//

#include "Physics_Tester.hh"
#include "Test_XS.hh"
#include "../Physics.cuh"
#include "gtest/Gtest_Functions.hh"
#include "cuda_geometry/Mesh_Geometry.hh"
#include "CudaUtils/cuda_utils/Shared_Device_Ptr.hh"
#include "Teuchos_RCP.hpp"
#include "Teuchos_ParameterList.hpp"

using namespace cuda_mc;

typedef cuda_profugus::Mesh_Geometry     Geom;
typedef cuda_profugus::Mesh_Geometry_DMM Geom_DMM;
typedef cuda_utils::Space_Vector         Space_Vector;
typedef Particle_Vector<Geom>            Particle_Vector_t;
typedef Particle_Vector_DMM<Geom>        Particle_Vector_DMM_t;

__global__ void test_total_kernel( Physics<Geom>    *phys,
                                   Particle_Vector_t particles,
                                   double           *totals,
                                   int               num_vals)
{
     int tid = threadIdx.x + blockIdx.x * blockDim.x;
     if( tid < num_vals )
     {
         int g = tid % 5;
         int matid = tid % 2;

         // Create particle
         particles.set_group(tid,g);
         particles.set_matid(tid,matid);
         totals[tid] = phys->total(profugus::physics::TOTAL,tid,particles);
     }
}

__global__ void test_collide_kernel( Geom               *geom,
                                     Physics<Geom>      *phys,
                                     Particle_Vector_t   particles,
                                     int                *events,
                                     int                 num_particles)
{
     int tid = threadIdx.x + blockIdx.x * blockDim.x;
     if( tid < num_particles )
     {
         int g = tid % 5;

         // Create particle
         particles.live(tid);
         particles.set_group(tid,g);
         particles.set_wt(tid,1.0);
         particles.set_event(tid,profugus::events::COLLISION);

         // Create and initialize RNG state
         hiprandState_t rng_state;
         hiprand_init(tid,0,0,&rng_state);
         particles.set_rng(tid,&rng_state);

         // Initialize geo state
         Space_Vector pos = {0.25, 0.75, 0.60};
         Space_Vector dir = {1.0, 0.0, 0.0};
         geom->initialize(pos,dir,particles.geo_state(tid));
         particles.set_matid(tid,geom->matid(particles.geo_state(tid)));

         // Collide
         phys->collide(tid,particles);

         events[tid] = particles.event(tid);

         printf("Particle %i has event %i, group %i, and weight %e\n",
            tid,particles.event(tid),particles.group(tid),particles.wt(tid));
     }
}

void Physics_Tester::test_total()
{
    auto xs = Test_XS::build_xs(5);

    int num_vals = 16;

    // Build geometry
    std::vector<double> edges = {0.0, 0.50, 1.0};
    std::vector<int> matids = {0, 1, 1, 0, 0, 1, 1, 0};
    auto geom_dmm = std::make_shared<Geom_DMM>(edges,edges,edges);
    geom_dmm->set_matids(matids);
    auto sdp_geom = cuda::shared_device_ptr<Geom>(geom_dmm->device_instance());

    // Build physics
    Teuchos::RCP<Teuchos::ParameterList> pl( new Teuchos::ParameterList() );
    auto sdp_mat = cuda::shared_device_ptr<cuda_profugus::XS_Device>(*xs);
    auto phys = std::make_shared<Physics<Geom> >(pl,xs,sdp_mat);
    phys->set_geometry(sdp_geom);
    auto sdp_phys = cuda::Shared_Device_Ptr<Physics<Geom> >(phys);

    // Build particles
    Particle_Vector_DMM_t particles;
    particles.initialize(num_vals);

    // Allocate data on device
    thrust::device_vector<double> device_totals(num_vals);

    test_total_kernel<<<1,num_vals>>>( sdp_phys.get_device_ptr(),
                                       particles.device_instance(),
                                       device_totals.data().get(),
                                       num_vals );

    REQUIRE( hipGetLastError() == hipSuccess );

    thrust::host_vector<double> host_totals = device_totals;

    for (int i = 0; i < num_vals; ++i)
    {
        int g = i % 5;
        int matid = i %2;
        const auto &expected = xs->vector(matid,profugus::XS::TOTAL);
        EXPECT_SOFT_EQ(expected[g], host_totals[i]);
    }
}

void Physics_Tester::test_collide()
{
    auto xs = Test_XS::build_xs(5);

    int num_particles = 16;

    // Build geometry
    std::vector<double> edges = {0.0, 0.50, 1.0};
    std::vector<int> matids = {0, 1, 1, 0, 0, 1, 1, 0};
    auto geom_dmm = std::make_shared<Geom_DMM>(edges,edges,edges);
    geom_dmm->set_matids(matids);
    auto sdp_geom = cuda::shared_device_ptr<Geom>(geom_dmm->device_instance());

    // Build physics
    Teuchos::RCP<Teuchos::ParameterList> pl( new Teuchos::ParameterList() );
    auto sdp_mat = cuda::shared_device_ptr<cuda_profugus::XS_Device>(*xs);

    auto phys = std::make_shared<Physics<Geom> >(pl,xs,sdp_mat);
    phys->set_geometry(sdp_geom);
    auto sdp_phys = cuda::Shared_Device_Ptr<Physics<Geom> >(phys);

    // Build particles
    Particle_Vector_DMM_t particles;
    particles.initialize(num_particles);

    thrust::device_vector<int> device_events(num_particles);

    test_collide_kernel<<<1,num_particles>>>( sdp_geom.get_device_ptr(),
                                              sdp_phys.get_device_ptr(),
                                              particles.device_instance(),
                                              device_events.data().get(),
                                              num_particles );

    EXPECT_EQ(hipGetLastError(), hipSuccess);

    thrust::host_vector<int> host_events = device_events;
    for (auto event : host_events)
        EXPECT_EQ(profugus::events::IMPLICIT_CAPTURE,event);

}

//---------------------------------------------------------------------------//
//                 end of Physics_Tester.cc
//---------------------------------------------------------------------------//
