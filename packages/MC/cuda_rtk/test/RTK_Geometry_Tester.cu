#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
/*!
 * \file   MC/cuda_rtk/test/RTK_Geometry_Tester.cu
 * \author Tom Evans
 * \date   Fri Feb 03 09:50:55 2017
 * \brief  RTK_Geometry_Tester member and kernel definitions.
 * \note   Copyright (c) 2017 Oak Ridge National Laboratory, UT-Battelle, LLC.
 */
//---------------------------------------------------------------------------//

#include <iomanip>
#include <cmath>
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "RTK_Geometry_Tester.hh"

#include "CudaUtils/cuda_utils/Utility_Functions.hh"
#include "CudaUtils/cuda_utils/Shared_Device_Ptr.hh"
#include "../RTK_Geometry.cuh"

//---------------------------------------------------------------------------//
// TYPES
//---------------------------------------------------------------------------//

using Device_Geometry         = cuda_profugus::Core;
using Device_Geometry_Manager = cuda_profugus::Core_DMM;
using State                   = Device_Geometry::Geo_State_t;
using Space_Vector            = Device_Geometry::Space_Vector;

//---------------------------------------------------------------------------//
// CORE
//---------------------------------------------------------------------------//

__global__
void heuristic_kernel(
    unsigned long   *dseeds,
    hiprandState_t   *rngs,
    Device_Geometry *geometry,
    int             *bins)
{
    // Get the thread id
    auto tid = cuda::utility::thread_id();

    // Initialize the random number states
    hiprand_init(dseeds[tid], 0, 0, rngs + tid);

    // Number of histories per thread
    int Np = 2000;

    // geometry variables
    double       costheta, sintheta, phi;
    Space_Vector r, omega;
    State        state;

    double two_pi = 6.283185307179586;

    // Sample Np tracks per thread
    for (int n = 0; n < Np; ++n)
    {
        // sample x,y,z randomly
        r[0] = hiprand_uniform_double(rngs + tid) * 7.56;
        r[1] = hiprand_uniform_double(rngs + tid) * 7.56;
        r[2] = hiprand_uniform_double(rngs + tid) * 28.56;

        // sample omega
        costheta = 1.0 - 2.0 * hiprand_uniform_double(rngs + tid);
        phi      = two_pi * hiprand_uniform_double(rngs + tid);
        sintheta = std::sqrt(1.0 - costheta * costheta);

        omega[0] = sintheta * cos(phi);
        omega[1] = sintheta * sin(phi);
        omega[2] = costheta;

        // initialize track
        geometry->initialize(r, omega, state);

        while (geometry->boundary_state(state) == profugus::geometry::INSIDE)
        {
            // get distance-to-boundary
            geometry->distance_to_boundary(state);

            // update position of particle and cross the surface
            geometry->move_to_surface(state);
        }

        if (state.escaping_face == State::MINUS_X)
            ++bins[0 + 6 * tid];
        else if (state.escaping_face == State::PLUS_X)
            ++bins[1 + 6 * tid];
        else if (state.escaping_face == State::MINUS_Y)
            ++bins[2 + 6 * tid];
        else if (state.escaping_face == State::PLUS_Y)
            ++bins[3 + 6 * tid];
        else if (state.escaping_face == State::MINUS_Z)
            ++bins[4 + 6 * tid];
        else if (state.escaping_face == State::PLUS_Z)
            ++bins[5 + 6 * tid];
    }
}

//---------------------------------------------------------------------------//

__global__
void reflecting_kernel(
    unsigned long   *dseeds,
    hiprandState_t   *rngs,
    Device_Geometry *geometry,
    int             *face_bin,
    int             *refl_bin)
{
    // Get the thread id
    auto tid = cuda::utility::thread_id();

    // Initialize the random number states
    hiprand_init(dseeds[tid], 0, 0, rngs + tid);

    // Number of histories per thread
    int Np = 2000;

    // geometry variables
    double       costheta, sintheta, phi;
    Space_Vector r, omega;
    State        state;

    double two_pi = 6.283185307179586;

    // sample Np tracks
    for (int n = 0; n < Np; ++n)
    {
        // sample x,y,z randomly
        r[0] = hiprand_uniform_double(rngs + tid) * 7.56;
        r[1] = hiprand_uniform_double(rngs + tid) * 7.56;
        r[2] = hiprand_uniform_double(rngs + tid) * 28.56;

        // sample omega
        costheta = 1.0 - 2.0 * hiprand_uniform_double(rngs + tid);
        phi      = two_pi * hiprand_uniform_double(rngs + tid);
        sintheta = sqrt(1.0 - costheta * costheta);

        omega[0] = sintheta * cos(phi);
        omega[1] = sintheta * sin(phi);
        omega[2] = costheta;

        // initialize track
        geometry->initialize(r, omega, state);

        // continue flag
        bool done = false;

        while (!done)
        {
            // get distance-to-boundary
            geometry->distance_to_boundary(state);

            // update position of particle to the surface and process it through
            geometry->move_to_surface(state);

            // if the particle is reflected then do the reflection
            if (geometry->boundary_state(state) == profugus::geometry::REFLECT)
            {
                if (state.exiting_face == State::MINUS_X)
                    ++refl_bin[0 + 6 * tid];
                else if (state.exiting_face == State::PLUS_X)
                    ++refl_bin[1 + 6 * tid];
                else if (state.exiting_face == State::MINUS_Y)
                    ++refl_bin[2 + 6 * tid];
                else if (state.exiting_face == State::PLUS_Y)
                    ++refl_bin[3 + 6 * tid];
                else if (state.exiting_face == State::MINUS_Z)
                    ++refl_bin[4 + 6 * tid];
                else if (state.exiting_face == State::PLUS_Z)
                    ++refl_bin[5 + 6 * tid];

                // reflect the particle
                geometry->reflect(state);
            }

            // terminate on escape
            if (geometry->boundary_state(state) == profugus::geometry::OUTSIDE)
            {
                done = true;
            }
        }

        if (state.escaping_face == State::MINUS_X)
            ++face_bin[0 + 6 * tid];
        else if (state.escaping_face == State::PLUS_X)
            ++face_bin[1 + 6 * tid];
        else if (state.escaping_face == State::MINUS_Y)
            ++face_bin[2 + 6 * tid];
        else if (state.escaping_face == State::PLUS_Y)
            ++face_bin[3 + 6 * tid];
        else if (state.escaping_face == State::MINUS_Z)
            ++face_bin[4 + 6 * tid];
        else if (state.escaping_face == State::PLUS_Z)
            ++face_bin[5 + 6 * tid];
    }
}

//---------------------------------------------------------------------------//

void Core::heuristic()
{
    using std::cout; using std::endl; using std::setw;

    build(false);

    // Build the manager
    Device_Geometry_Manager dmm(*geometry);

    // Get the host object
    auto device_geo = cuda::shared_device_ptr<Device_Geometry>(
        dmm.device_instance());

    // Send over random number seeds and states
    thrust::device_vector<unsigned long> dseeds(seeds.begin(), seeds.end());
    thrust::device_vector<hiprandState_t> rngs(num_threads * num_blocks);

    // Testing data
    thrust::device_vector<int> bins(num_threads * num_blocks * 6, 0);

    dim3 blocks  = {num_blocks};
    dim3 threads = {num_threads};
    heuristic_kernel<<<blocks,threads>>>(dseeds.data().get(),
                                         rngs.data().get(),
                                         device_geo.get_device_ptr(),
                                         bins.data().get());
    hipDeviceSynchronize();

    thrust::host_vector<int> rbins(bins.begin(), bins.end());

    // Sum up bin results
    std::vector<double> face_bin(6, 0.0);
    for (int t = 0; t < num_threads * num_blocks; ++t)
    {
        for (int b = 0; b < 6; ++b)
        {
            face_bin[b] += rbins[b + 6 * t];
        }
    }

    double Npx = static_cast<double>(2000 * num_threads * num_blocks);

    EXPECT_EQ(Npx, face_bin[0] + face_bin[1] + face_bin[2] +
              face_bin[3] + face_bin[4] + face_bin[5]);

    double xyf  = 28.56 * 7.56;
    double zf   = 7.56 * 7.56;
    double area = 4 * xyf + 2 * zf;
    double lox  = face_bin[0] / Npx;
    double hix  = face_bin[1] / Npx;
    double loy  = face_bin[2] / Npx;
    double hiy  = face_bin[3] / Npx;
    double loz  = face_bin[4] / Npx;
    double hiz  = face_bin[5] / Npx;

    EXPECT_SOFTEQ(lox, xyf / area, 0.005);
    EXPECT_SOFTEQ(hix, xyf / area, 0.005);
    EXPECT_SOFTEQ(loy, xyf / area, 0.005);
    EXPECT_SOFTEQ(hiy, xyf / area, 0.005);
    EXPECT_SOFTEQ(loz, zf / area,  0.030);
    EXPECT_SOFTEQ(hiz, zf / area,  0.030);

    cout.precision(6);
    cout << std::fixed << endl;
    cout << "Low  X leakage = "
         << setw(10) << lox << " ("
         << setw(10) << xyf / area << ")" << endl;
    cout << "High X leakage = "
         << setw(10) << hix << " ("
         << setw(10) << xyf / area << ")" << endl;
    cout << "Low  Y leakage = "
         << setw(10) << loy << " ("
         << setw(10) << xyf / area << ")" << endl;
    cout << "High Y leakage = "
         << setw(10) << hiy << " ("
         << setw(10) << xyf / area << ")" << endl;
    cout << "Low  Z leakage = "
         << setw(10) << loz << " ("
         << setw(10) << zf / area << ")" << endl;
    cout << "High Z leakage = "
         << setw(10) << hiz << " ("
         << setw(10) << zf / area << ")" << endl;
    cout << endl;
}

//---------------------------------------------------------------------------//

void Core::reflecting()
{
    using std::cout; using std::endl; using std::setw;

    build(true);

    // Build the manager
    Device_Geometry_Manager dmm(*geometry);

    // Get the host object
    auto device_geo = cuda::shared_device_ptr<Device_Geometry>(
        dmm.device_instance());

    // Send over random number seeds and states
    thrust::device_vector<unsigned long> dseeds(seeds.begin(), seeds.end());
    thrust::device_vector<hiprandState_t> rngs(num_threads * num_blocks);

    // Testing data
    thrust::device_vector<int> face_bins(num_threads * num_blocks * 6, 0);
    thrust::device_vector<int> refl_bins(num_threads * num_blocks * 6, 0);

    dim3 blocks  = {num_blocks};
    dim3 threads = {num_threads};
    reflecting_kernel<<<blocks,threads>>>(dseeds.data().get(),
                                          rngs.data().get(),
                                          device_geo.get_device_ptr(),
                                          face_bins.data().get(),
                                          refl_bins.data().get());

    thrust::host_vector<int> rface(face_bins.begin(), face_bins.end());
    thrust::host_vector<int> rrefl(refl_bins.begin(), refl_bins.end());

    // Sum up bin results
    std::vector<double> face_bin(6, 0.0);
    std::vector<double> refl_bin(6, 0.0);
    for (int t = 0; t < num_threads * num_blocks; ++t)
    {
        for (int b = 0; b < 6; ++b)
        {
            face_bin[b] += rface[b + 6 * t];
            refl_bin[b] += rrefl[b + 6 * t];
        }
    }

    double Npx = static_cast<double>(2000 * num_threads * num_blocks);

    EXPECT_EQ(Npx, face_bin[0] + face_bin[1] + face_bin[2] +
              face_bin[3] + face_bin[4] + face_bin[5]);

    EXPECT_EQ(0, face_bin[0]);
    EXPECT_EQ(0, refl_bin[1]);
    EXPECT_EQ(0, face_bin[2]);
    EXPECT_EQ(0, refl_bin[3]);
    EXPECT_EQ(0, face_bin[4]);
    EXPECT_EQ(0, refl_bin[5]);

    double xyf  = 28.56 * 7.56;
    double zf   = 7.56 * 7.56;
    double area = 2 * xyf + 1 * zf;
    double hix  = face_bin[1] / Npx;
    double hiy  = face_bin[3] / Npx;
    double hiz  = face_bin[5] / Npx;

    EXPECT_SOFTEQ(hix, xyf / area, 0.005);
    EXPECT_SOFTEQ(hiy, xyf / area, 0.005);
    EXPECT_SOFTEQ(hiz, zf / area,  0.030);

    cout.precision(6);
    cout << std::fixed << endl;
    cout << "High X leakage = "
         << setw(8) << hix << " ("
         << setw(8) << xyf / area << ")" << endl;
    cout << "High Y leakage = "
         << setw(8) << hiy << " ("
         << setw(8) << xyf / area << ")" << endl;
    cout << "High Z leakage = "
         << setw(8) << hiz << " ("
         << setw(8) << zf / area << ")" << endl;

    cout << std::scientific << endl;

    cout.precision(2);
    cout << "Low  X reflection = " << setw(8) << refl_bin[0] << endl;
    cout << "Low  Y reflection = " << setw(8) << refl_bin[2] << endl;
    cout << "Low  Z reflection = " << setw(8) << refl_bin[4] << endl;
    cout << endl;
}

//---------------------------------------------------------------------------//
// end of MC/cuda_rtk/test/RTK_Geometry_Tester.cu
//---------------------------------------------------------------------------//
