#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
/*!
 * \file   MC/cuda_rtk/RTK_Cell.cu
 * \author Tom Evans
 * \date   Mon Nov 28 12:33:05 2016
 * \brief  RTK_Cell kernel definitions.
 * \note   Copyright (c) 2016 Oak Ridge National Laboratory, UT-Battelle, LLC.
 */
//---------------------------------------------------------------------------//

#include <cmath>

#include "RTK_Cell.cuh"

namespace cuda_profugus
{

//---------------------------------------------------------------------------//
// RTK_CELL MEMBERS
//---------------------------------------------------------------------------//
/*!
 * \brief Constructor.
 */
RTK_Cell::RTK_Cell(int      mod_id,
                   View_Dbl r,
                   View_Int ids,
                   double   (&extents)[2][2],
                   double   height,
                   int      num_segments)
    : d_mod_id(mod_id)
    , d_r(r)
    , d_ids(ids)
    , d_z(height)
    , d_num_shells(r.size())
    , d_num_regions(d_num_shells + 1)
    , d_segments(num_segments)
    , d_seg_faces(d_segments / 2)
    , d_num_int_faces(d_seg_faces + d_num_shells)
    , d_mod_region(d_num_regions - 1)
    , d_num_cells(d_num_regions * d_segments)
    , d_vessel(false)
{
    using def::X; using def::Y;

    // Set the extents
    d_extent[X][LO] = extents[X][LO];
    d_extent[X][HI] = extents[X][HI];
    d_extent[Y][LO] = extents[Y][LO];
    d_extent[Y][HI] = extents[Y][HI];

    // Set the pitch
    d_xy[X] = d_extent[X][HI] - d_extent[X][LO];
    d_xy[Y] = d_extent[Y][HI] - d_extent[Y][LO];
}

//---------------------------------------------------------------------------//
/*!
 * \brief Add a vessel.
 */
void RTK_Cell::add_vessel(
    int    vessel_id,
    double R0,
    double R1,
    double (&offsets)[2])
{
    using def::X; using def::Y;

    // Set internal data
    d_vessel     = true;
    d_vessel_id  = vessel_id;
    d_offsets[0] = offsets[0];
    d_offsets[1] = offsets[1];
    d_R0         = -1.0;
    d_R1         = -1.0;
    d_inner      = false;
    d_outer      = false;

    // Near and far corners relative to the origin of vessel cylinder
    double near[2], far[2];

    for (int dir = 0; dir < 2; ++dir)
    {
        near[dir] = this->l2g(d_extent[dir][LO], dir);
        far[dir]  = this->l2g(d_extent[dir][HI], dir);

        if (d_offsets[dir] < 0.0)
        {
            near[dir] = this->l2g(d_extent[dir][HI], dir);
            far[dir]  = this->l2g(d_extent[dir][LO], dir);
        }
    }

    // calculate the near and far radii bisecting the pincell
    double nearR2 = near[X]*near[X] + near[Y]*near[Y];
    double farR2  = far[X]*far[X] + far[Y]*far[Y];
    double R0_2   = R0 * R0;
    double R1_2   = R1 * R1;

    // check to see if R0 or R1 bisect the cell, R0 < R1 so if R0 > farR the
    // vessel does not bisect the cell
    VALIDATE(R0_2 < farR2,
             "R0 = " << R0 << " is greater than the far extent "
             << "of the pincell, " << std::sqrt(farR2));

    // likewise if R1 < nearR the vessel cannot bisect the cell
    VALIDATE(R1_2 > nearR2,
             "R1 = " << R1 <<  " is less than the near extent "
             << "of the pincell, " << std::sqrt(farR2));

    // now we have to check each vessel radius
    if (R0_2 > nearR2)
    {
        CHECK(R0_2 < farR2);
        d_R0    = R0;
        d_inner = true;
    }

    if (R1_2 < farR2)
    {
        CHECK(R1_2 > nearR2);
        d_R1    = R1;
        d_outer = true;
    }
}

//---------------------------------------------------------------------------//
// RTK_CELL_DMM MEMBERS
//---------------------------------------------------------------------------//
/*!
 * \brief Constructor.
 */
RTK_Cell_DMM::RTK_Cell_DMM(
    const Host_RTK_Cell &host_cell)
    : d_num_segments(host_cell.num_segments())
    , d_z(host_cell.height())
{
    using def::X; using def::Y;

    // Number of shells in this cell
    auto num_shells = host_cell.num_shells();

    // Calculate radii and ids
    if (num_shells > 0)
    {
        // Radii
        const auto &radii = host_cell.radii();
        CHECK(!radii.empty());

        d_r = thrust::device_vector<double>(radii.begin(), radii.end());

        // IDS
        std::vector<int> ids(num_shells);
        for (int n = 0; n < num_shells; ++n)
        {
            ids[n] = host_cell.matid(n);
        }
        d_ids = thrust::device_vector<int>(ids.begin(), ids.end());
    }

    // Number of regions in this cell
    auto num_regions = host_cell.num_regions();
    CHECK(num_regions == num_shells + 1);

    // Store the moderator id
    d_mod_id = host_cell.matid(num_regions - 1);

    // Get the extents
    Host_RTK_Cell::Space_Vector low, high;
    host_cell.get_extents(low, high);
    d_extent[X][0] = low[X];
    d_extent[X][1] = high[X];
    d_extent[Y][0] = low[Y];
    d_extent[Y][1] = high[Y];
    CHECK(high[2] == d_z);

    // Store vessel data
    d_vessel = host_cell.vessel_data(
        d_vessel_id, d_R0, d_R1, d_offsets[0], d_offsets[1]);

    ENSURE(host_cell.num_cells() == num_regions * d_num_segments);
}

//---------------------------------------------------------------------------//
/*!
 * \brief Construct a device RTK_Cell object.
 */
RTK_Cell RTK_Cell_DMM::device_instance()
{
    RTK_Cell cell(d_mod_id,
                  cuda::make_view(d_r),
                  cuda::make_view(d_ids),
                  d_extent,
                  d_z,
                  d_num_segments);

    // Add the vessel if it exists
    if (d_vessel) cell.add_vessel(d_vessel_id, d_R0, d_R1, d_offsets);

    return cell;
}

} // end namespace cuda_profugus

//---------------------------------------------------------------------------//
// end of MC/cuda_rtk/RTK_Cell.cu
//---------------------------------------------------------------------------//
